#include "hip/hip_runtime.h"
#include "ahpg.cuh"

AHPG::AHPG() {
	this->hierarchy = new Hierarchy<VariablesType, WeightType>();
	IR[3] = 0.5245;
	IR[4] = 0.8815;
	IR[5] = 1.1086;
	IR[6] = 1.1279;
	IR[7] = 1.3417;
	IR[8] = 1.4056;
	IR[9] = 1.4499;
	IR[10] = 1.4854;
	IR[11] = 1.5141;
	IR[12] = 1.5365;
	IR[13] = 1.5551;
	IR[14] = 1.5713;
	IR[15] = 1.5838;
}

std::string AHPG::strToLowerG(std::string s) {
	std::transform(s.begin(), s.end(), s.begin(),
	               [](unsigned char c) {
		return std::tolower(c);
	});
	return s;
}

void AHPG::updateAlternativesG() {
	this->hierarchy->clearAlternatives();
	rapidjson::SchemaDocument alternativesSchema =
		JSON::generateSchema("multicriteria/json/alternativesSchema.json");
	rapidjson::Document alternativesData =
		JSON::generateDocument("multicriteria/json/alternativesDataDefault.json");
	rapidjson::SchemaValidator alternativesValidator(alternativesSchema);
	if (!alternativesData.Accept(alternativesValidator))
		JSON::jsonError(&alternativesValidator);
	domParserG(&alternativesData);
	this->hierarchy->addEdgeSheetsAlternatives();
}

// Recieve a function address to iterate, used in build matrix, normalized, pml
// and pg functions.
template <typename F, typename T> void AHPG::iterateFuncG(F f, T *v) {
	std::vector<Hierarchy<VariablesType, WeightType>::Edge *> e = v->getEdges();
	Hierarchy<VariablesType, WeightType>::Criteria *c;
	for (edgeIt it = e.begin(); it != e.end(); it++) {
		c = (*it)->getCriteria();
		if (c != NULL) {
			(this->*f)(c);
		}
	}
}

template <typename T> void AHPG::buildMatrixG(T *v) {
	int size = v->edgesCount();
	WeightType **matrix = new (std::nothrow) WeightType *[size];
	for (int i = 0; i < size; i++)
		matrix[i] = new (std::nothrow) WeightType[size];
	std::vector<WeightType> w;
	for (int i = 0; i < size; i++) {
		matrix[i][i] = 1;
		w = (v->getEdges())[i]->getWeights();
		for (int j = i + 1; j < size; j++) {
			matrix[i][j] = w[j];
			matrix[j][i] = 1 / matrix[i][j];
		}
	}
	v->setMatrix(matrix);
	iterateFuncG(
		&AHPG::buildMatrixG<Hierarchy<VariablesType, WeightType>::Criteria>, v);
}

template <typename T> void AHPG::buildNormalizedmatrixG(T *v) {
	int size = v->edgesCount();
	WeightType **matrix = v->getMatrix(), sum = 0;
	WeightType **nMatrix = new (std::nothrow) WeightType *[size];
	for (int i = 0; i < size; i++)
		nMatrix[i] = new (std::nothrow) WeightType[size];
	for (int i = 0; i < size; i++) {
		sum = 0;
		for (int j = 0; j < size; j++) {
			sum += matrix[j][i];
		}
		for (int j = 0; j < size; j++) {
			nMatrix[j][i] = matrix[j][i] / sum;
		}
	}
	v->setNormalizedMatrix(nMatrix);
	iterateFuncG(&AHPG::buildNormalizedmatrixG<
			     Hierarchy<VariablesType, WeightType>::Criteria>,
	             v);
}

template <typename T> void AHPG::buildPmlG(T *v) {
	int size = v->edgesCount();
	WeightType sum = 0;
	WeightType *pml = new (std::nothrow) WeightType[size];
	WeightType **matrix = v->getNormalizedMatrix();
	for (int i = 0; i < size; i++) {
		sum = 0;
		for (int j = 0; j < size; j++) {
			sum += matrix[i][j];
		}
		pml[i] = sum / size;
	}
	v->setPml(pml);
	iterateFuncG(&AHPG::buildPmlG<Hierarchy<VariablesType, WeightType>::Criteria>,
	             v);
}

template <typename T> void AHPG::buildPgG(T *v) {
	int aSize = this->hierarchy->getAlternativesCount();
	std::vector<Hierarchy<VariablesType, WeightType>::Edge *> e = v->getEdges();
	WeightType *pg = new (std::nothrow) WeightType[aSize];
	for (int i = 0; i < aSize; i++) {
		pg[i] = partialPgG(v, i);
	}
	v->setPg(pg);
}

template <typename T> WeightType AHPG::partialPgG(T *v, int alternative) {
	std::vector<Hierarchy<VariablesType, WeightType>::Edge *> e = v->getEdges();
	int size = e.size();
	Hierarchy<VariablesType, WeightType>::Criteria *c;
	WeightType *pml = v->getPml();
	WeightType partial = 0;
	for (int i = 0; i < size; i++) {
		c = e[i]->getCriteria();
		if (c != NULL) {
			partial += pml[i] * partialPgG(c, alternative);
		} else {
			return pml[alternative];
		}
	}
	return partial;
}

template <typename T> void AHPG::deleteMatrixG(T *v) {
	int size = v->edgesCount();
	WeightType **matrix = v->getMatrix();
	for (int i = 0; i < size; i++)
		delete[] matrix[i];
	delete[] matrix;
	matrix = NULL;
	v->setMatrix(NULL);
	iterateFuncG(
		&AHPG::deleteMatrixG<Hierarchy<VariablesType, WeightType>::Criteria>, v);
}

template <typename T> void AHPG::deleteNormalizedMatrixG(T *v) {
	int size = v->edgesCount();
	WeightType **nMatrix = v->getNormalizedMatrix();
	for (int i = 0; i < size; i++)
		delete[] nMatrix[i];
	delete[] nMatrix;
	nMatrix = NULL;
	v->setNormalizedMatrix(NULL);
	iterateFuncG(&AHPG::deleteNormalizedMatrixG<
			     Hierarchy<VariablesType, WeightType>::Criteria>,
	             v);
}

template <typename T> void AHPG::checkConsistencyG(T *v) {
	int size = v->edgesCount();
	WeightType **matrix = v->getMatrix();
	WeightType *pml = v->getPml();
	WeightType p[size], lambda = 0, RC = 0;
	for (int i = 0; i < size; i++) {
		p[i] = 0;
		for (int j = 0; j < size; j++) {
			p[i] += pml[j] * matrix[i][j];
		}
		lambda += (p[i] / pml[i]);
	}
	lambda /= size;
	if (IR[size] > 0) {
		RC = (abs(lambda - size) / (size - 1)) / IR[size];
	} else {
		// according to AlonsoLamata 2006
		// RC = CI/ RI , where
		// CI = (Lambda_max - n ) / (n-1), and
		// RI = (~Lambda_max - n) / (n-1) , so
		// RC = (Lambda_max - n) / (n-1) / (~Lambda_max - n) / (n-1), then
		// RC = (Lambda_max - n) / (~Lambda_max - n), the ~Lambda_max can be
		// calculated through ~Lambda_max = 2.7699*n-4.3513, thus RC = (Lambda_max -
		// n) / (2.7699 * n - 4.3513 - n ), simplifying RC = (Lambda_max - n) /
		// (1.7699 * n - 4.3513)
		RC = (fabs(lambda - size) / (1.7699 * size - 4.3513));
	}
	if (RC > 0.1) {
		std::cout << "ERROR: Criteria: " << v->getName() << " is inconsistent\n";
		std::cout << "RC= " << RC << "\n";
		std::cout<<"SIZE: "<<size<<"\n";
		printMatrixG(v);
		printNormalizedMatrixG(v);
		printPmlG(v);
		exit(0);
	}
	iterateFuncG(
		&AHPG::checkConsistencyG<Hierarchy<VariablesType, WeightType>::Criteria>,
		v);
}

void AHPG::generateContentSchemaG() {
	std::string names;
	std::string text = "{\"$schema\":\"http://json-schema.org/draft-04/"
	                   "schema#\",\"definitions\": {\"alternative\": {\"type\": "
	                   "\"array\",\"minItems\": 1,\"items\":{\"properties\": {";
	auto resource = this->hierarchy->getResource();
	for (auto it : resource->mInt) {
		text += "\"" + it.first + "\":{\"type\":\"number\"},";
		names += "\"" + it.first + "\",";
	}
	for (auto it : resource->mWeight) {
		text += "\"" + it.first + "\":{\"type\":\"number\"},";
		names += "\"" + it.first + "\",";
	}
	for (auto it : resource->mBool) {
		text += "\"" + it.first + "\":{\"type\":\"boolean\"},";
		names += "\"" + it.first + "\",";
	}
	for (auto it : resource->mString) {
		text += "\"" + it.first + "\":{\"type\":\"string\"},";
		names += "\"" + it.first + "\",";
	}
	names.pop_back();
	text.pop_back();
	text += "},\"additionalProperties\": false,\"required\": [" + names +
	        "]}}},\"type\": \"object\",\"minProperties\": "
	        "1,\"additionalProperties\": false,\"properties\": "
	        "{\"alternatives\": {\"$ref\": \"#/definitions/alternative\"}}}";
	JSON::writeJson("multicriteria/json/alternativesSchema.json", text);
}

template <typename T> void AHPG::printMatrixG(T *v) {
	WeightType **matrix = v->getMatrix();
	int tam = v->edgesCount();
	std::cout << "Matrix of " << v->getName() << "\n";
	for (int i = 0; i < tam; i++) {
		for (int j = 0; j < tam; j++) {
			std::cout << std::setfill(' ') << std::setw(10) << matrix[i][j] << " ";
		}
		std::cout << "\n";
	}
	std::cout << "\n";
	iterateFuncG(
		&AHPG::printMatrixG<Hierarchy<VariablesType, WeightType>::Criteria>, v);
}

template <typename T> void AHPG::printNormalizedMatrixG(T *v) {
	WeightType **matrix = v->getNormalizedMatrix();
	int tam = v->edgesCount();
	std::cout << "Normalized Matrix of " << v->getName() << "\n";
	for (int i = 0; i < tam; i++) {
		for (int j = 0; j < tam; j++) {
			std::cout << std::setfill(' ') << std::setw(10) << matrix[i][j] << " ";
		}
		std::cout << "\n";
	}
	std::cout << "\n";
	iterateFuncG(&AHPG::printNormalizedMatrixG<
			     Hierarchy<VariablesType, WeightType>::Criteria>,
	             v);
}

template <typename T> void AHPG::printPmlG(T *v) {
	WeightType *pml = v->getPml();
	int tam = v->edgesCount();
	std::cout << "PML of " << v->getName() << "\n";
	for (int i = 0; i < tam; i++) {
		std::cout << std::setfill(' ') << std::setw(10) << pml[i] << " ";
	}
	std::cout << "\n";
	iterateFuncG(&AHPG::printPmlG<Hierarchy<VariablesType, WeightType>::Criteria>,
	             v);
}

template <typename T> void AHPG::printPgG(T *v) {
	WeightType *pg = v->getPg();
	int tam = this->hierarchy->getAlternativesCount();
	std::cout << "PG of " << v->getName() << "\n";
	for (int i = 0; i < tam; i++) {
		std::cout << std::setfill(' ') << std::setw(10) << pg[i] << " ";
	}
	std::cout << "\n";
}

void AHPG::resourcesParserG(genericValue *dataResource) {
	std::string variableName, variableType;
	for (auto &arrayData : dataResource->value.GetArray()) {
		variableName = variableType = "";
		for (auto &objectData : arrayData.GetObject()) {
			if (strcmp(objectData.name.GetString(), "name") == 0) {
				variableName = objectData.value.GetString();
			} else if (strcmp(objectData.name.GetString(), "variableType") == 0) {
				variableType = strToLowerG(objectData.value.GetString());
			} else {
				std::cout << "Error in reading resources\nExiting...\n";
				exit(0);
			}
		}
		this->hierarchy->addResource(variableName, variableType);
	}
}

void AHPG::hierarchyParserG(genericValue *dataObjective) {
	for (auto &hierarchyObject : dataObjective->value.GetObject()) {
		if (strcmp(hierarchyObject.name.GetString(), "name") == 0) {
			this->hierarchy->addFocus(
				strToLowerG(hierarchyObject.value
				            .GetString())); // create the Focus* in the hierarchy;
		} else if (strcmp(hierarchyObject.name.GetString(), "childs") == 0) {
			criteriasParserG(&hierarchyObject, this->hierarchy->getFocus());
		} else {
			std::cout << "AHPG -> Unrecognizable Type\nExiting...\n";
			exit(0);
		}
	}
}

template <typename Parent>
void AHPG::criteriasParserG(genericValue *dataCriteria, Parent p) {
	std::string name = " ";
	bool leaf = false;
	std::vector<WeightType> weight;
	for (auto &childArray : dataCriteria->value.GetArray()) {
		weight.clear();
		for (auto &child : childArray.GetObject()) {
			const char *n = child.name.GetString();
			if (strcmp(n, "name") == 0) {
				name = strToLowerG(child.value.GetString());
			} else if (strcmp(n, "leaf") == 0) {
				leaf = child.value.GetBool();
			} else if (strcmp(n, "weight") == 0) {
				for (auto &weightChild : child.value.GetArray()) {
					weight.push_back(weightChild.GetDouble());
				}
			} else if (strcmp(n, "childs") == 0) {
				// at this point, all the criteria variables were read, now the document
				// has the child's of the criteria. To put the childs corretly inside
				// the hierarchy, the criteria node has to be created.
				auto criteria = this->hierarchy->addCriteria(name);
				criteria->setLeaf(leaf);
				this->hierarchy->addEdge(p, criteria, weight);
				// with the criteria node added, the call recursively the
				// criteriasParser.
				criteriasParserG(&child, criteria);
			}
		}
		if (leaf) {
			auto criteria = this->hierarchy->addCriteria(name);
			criteria->setLeaf(leaf);
			this->hierarchy->addSheets(criteria);
			this->hierarchy->addEdge(p, criteria, weight);
		}
	}
}

void AHPG::alternativesParserG(genericValue *dataAlternative) {
	for (auto &arrayAlternative : dataAlternative->value.GetArray()) {
		auto alternative = this->hierarchy->addAlternative();
		for (auto &alt : arrayAlternative.GetObject()) {
			std::string name(alt.name.GetString());
			if (alt.value.IsNumber()) {
				if (alternative->getResource()->mInt.count(name) > 0) {
					alternative->setResource(name, alt.value.GetInt());
				} else {
					alternative->setResource(name, alt.value.GetFloat());
				}
			} else if (alt.value.IsBool()) {
				alternative->setResource(name, alt.value.GetBool());
			} else {
				alternative->setResource(
					name, strToLowerG(std::string(alt.value.GetString())));
			}
		}
	}
}

void AHPG::domParserG(rapidjson::Document *data) {
	for (auto &m : data->GetObject()) { // query through all objects in data.
		if (strcmp(m.name.GetString(), "resources") == 0) {
			resourcesParserG(&m);
		} else if (strcmp(m.name.GetString(), "objective") == 0) {
			hierarchyParserG(&m);
		} else if (strcmp(m.name.GetString(), "alternatives") == 0) {
			alternativesParserG(&m);
		}
	}
}

void AHPG::conceptionG(bool alternativeParser) {
	// The hierarchy contruction were divided in three parts, first the resources
	// file was to be loaded to construct the alternatives dynamically. Second the
	// hierarchy focus and criteria were loaded in the hierarchyData.json, and
	// finally the alternatives were loaded.
	if (alternativeParser) {
		rapidjson::SchemaDocument resourcesSchema =
			JSON::generateSchema("multicriteria/json/resourcesSchema.json");
		rapidjson::Document resourcesData =
			JSON::generateDocument("multicriteria/json/resourcesData.json");
		rapidjson::SchemaValidator resourcesValidator(resourcesSchema);
		if (!resourcesData.Accept(resourcesValidator))
			JSON::jsonError(&resourcesValidator);
		domParserG(&resourcesData);
		generateContentSchemaG();
	}
	// After reading the resoucesData, new alternativesSchema has to be created.
	// Parser the Json File that contains the Hierarchy
	rapidjson::SchemaDocument hierarchySchema =
		JSON::generateSchema("multicriteria/json/hierarchySchema.json");
	rapidjson::Document hierarchyData =
		JSON::generateDocument("multicriteria/json/hierarchyData.json");
	rapidjson::SchemaValidator hierarchyValidator(hierarchySchema);
	if (!hierarchyData.Accept(hierarchyValidator))
		JSON::jsonError(&hierarchyValidator);
	domParserG(&hierarchyData);
	if (alternativeParser) {
		// The Json Data is valid and can be used to construct the hierarchy.
		rapidjson::SchemaDocument alternativesSchema =
			JSON::generateSchema("multicriteria/json/alternativesSchema.json");
		rapidjson::Document alternativesData = JSON::generateDocument(
			"multicriteria/json/alternativesDataDefault.json");
		rapidjson::SchemaValidator alternativesValidator(alternativesSchema);
		if (!alternativesData.Accept(alternativesValidator))
			JSON::jsonError(&alternativesValidator);
		domParserG(&alternativesData);
		this->hierarchy->addEdgeSheetsAlternatives();
	}
}

__device__
int char_to_int(char* str){
	int res = 0; // Initialize result
	int sign = 1;  // Initialize sign as positive
	int i = 0;  // Initialize index of first digit
	// If number is negative, then update sign
	if (str[0] == '-') {
		sign = -1;
		i++; // Also update index of first digit
	}
	// Iterate through all digits and update the result
	for (; str[i] != '\0'; ++i)
		res = res*10 + str[i] - '0';
	// Return result with sign
	return sign*res;
}

__device__
float char_to_float(char* str){
	float rez = 0, fact = 1;
	if (*str == '-') {
		str++;
		fact = -1;
	};
	for (int point_seen = 0; *str; str++) {
		if (*str == '.') {
			point_seen = 1;
			continue;
		};
		int d = *str - '0';
		if (d >= 0 && d <= 9) {
			if (point_seen) fact /= 10.0f;
			rez = rez * 10.0f + (float)d;
		};
	}
	// printf("%f AND %f\n",rez,fact);
	return rez*fact;
}

__device__
char* copyStr(char* data, int init, int end){
	char* str = (char*)malloc(20);
	int i,k;
	for(i=init,k=0; i<end; i++,k++)
		str[k]=data[i];
	str[k]='\0';
	return str;
}

/*
    This function is used to make the comparison between the alternatives.
    Data represents all the data in the alternatives.
    Types represents the types of the alternative data.
    Max_mix represents to normalize the values.
    cmp is the result array of the comparison.
    size is the amount of the alternatives in the hierarchy.
    sizeCrit represent the size of sheets, used to jump through the data and types array.
 */
__global__
void acquisitonGKernel(char * data, int* index,  int* types, float* max_min, float* cmp, int size, int sizeCrit){
	//The row is the number of the alternative
	int row = blockIdx.x*size+threadIdx.x;
	//The col is the number of the criteria
	int col = blockIdx.y*size+threadIdx.y;
	int value_alt1_int, value_alt2_int, t;
	float value_alt1_float, value_alt2_float;
	char* sub,*sub2;
	// int k=0;
	// for(int i=0; data[i]!='\0'; i++) {
	//      if(i==index[k]) {printf("|"); k++;}
	//      printf("%c",data[i]);
	// }
	// printf("ALT SIZE: %d , CRIT SIZE %d\n",size, sizeCrit);
	// if(row==0 && col <sizeCrit) { //the thread can do the work
	if(row<size && col <sizeCrit) {                 //the thread can do the work
		int indexRead = row*sizeCrit+col;
		// printf("NEW THREAD ROW %d COL %d SIZE %d SIZECRIT %d\n",row,col,size,sizeCrit);
		value_alt1_int=0;
		value_alt1_float=0.0f;
		t=0;
		// printf("%d # %d # %d # %d\n",indexRead, indexRead+1,index[indexRead],index[indexRead+1]);
		sub=copyStr(data,index[indexRead],index[indexRead+1]);
		if(types[row*sizeCrit+col]==0 || types[row*sizeCrit+col]==2) {
			value_alt1_int=char_to_int(sub);
			// printf("CONVERTED INT %d\n",value_alt1_int);
		}else if(types[row*sizeCrit+col]==1) {
			value_alt1_float=char_to_float(sub);
			// printf("CONVERTED FLOAT %f\n",value_alt1_float);
		}
		for(int alt=0; alt<size; alt++) {
			sub2=copyStr(data,index[alt*sizeCrit+col],index[alt*sizeCrit+(col+1)]);
			// printf("ALTERNATIVE %d - %s # %s\n",alt,sub,sub2);
			value_alt2_int=0;
			value_alt2_float=0.0f;
			//alt*sizeCrit+col will jump over the alternatives to get the same coleria value.
			if(types[alt*sizeCrit+col]==0) {
				t=0;
				value_alt2_int=char_to_int(sub2);
			}else if(types[alt*sizeCrit+col]==1) {
				t=1;
				value_alt2_float=char_to_float(sub2);
			}else if(types[alt*sizeCrit+col]==2) {
				t=2;
				value_alt2_int=char_to_int(sub2);
			}
			// printf("DIVIDED BY %f\n",max_min[col]);
			// printf("SIZE: %d\n",size);
			int indexWrite = row*size*sizeCrit+col*size+alt;
			// int indexWrite = row*size*size/2+alt;
			// printf("WERE I WRITE %d\n",row*size*sizeCrit+col*size+alt);
			//Its used row*size*size/2 to jump correctly in the vector and set the values
			if(t==0) {
				value_alt1_int==value_alt2_int ? cmp[indexWrite]=1 : cmp[indexWrite] = (value_alt1_int-value_alt2_int) / (float) max_min[col];
				// printf("Write in T0 %f\n",cmp[indexWrite]);
			}else if(t==1) {
				// if(value_alt1_float!=value_alt2_float) printf("DIF %f %f\n",value_alt1_float,value_alt2_float);
				value_alt1_float==value_alt2_float ? cmp[indexWrite]=1 : cmp[indexWrite] = (value_alt1_float - value_alt2_float) / (float) max_min[col];
				// printf("Write in T1 %f\n",cmp[indexWrite]);
			}
			else if(t==2) {
				// printf("ALTERNATIVE %d - %s # %s\n",alt,sub,sub2);
				// printf("BOOL %d %d\n",value_alt1_int,value_alt2_int);
				if(value_alt1_int==value_alt2_int) cmp[indexWrite]=1;
				else if(value_alt1_int==1) cmp[indexWrite]=9;
				else if(value_alt1_int==0) cmp[indexWrite]=1/9.0f;
				// printf("Write in T2 %f\n", cmp[indexWrite]);
			}else{
				printf("UNESPECTED VALUE FOR T\n");
			}
		}
	}
}

void AHPG::acquisitionG() {
	//Get the device info
	int devID;
	hipDeviceProp_t props;
	hipGetDevice(&devID);
	hipGetDeviceProperties(&props, devID);
	int block_size = (props.major < 2) ? 16 : 32;

	// Para gerar os pesos das alterntivas, será primeiro captado o MIN e MAX
	// valor das alternativas , após isso será montada as matrizes de cada sheet
	auto alt = this->hierarchy->getAlternatives();
	auto sheets = this->hierarchy->getSheets();
	std::vector<std::string> sheetsNames;
	for_each(sheets.begin(), sheets.end(),
	         [&sheetsNames,
	          this](Hierarchy<VariablesType, WeightType>::Criteria *c) mutable {
		// Get the name off all sheet nodes that aren't boolean
		if (this->hierarchy->getResource()->mBool.count(c->getName()) ==
		    0) {
		        sheetsNames.push_back(c->getName());
		}
	});
	std::map<std::string, float> resultValues;
	std::vector<float>h_resources;
	float min, max;
	for (auto it = sheetsNames.begin(); it != sheetsNames.end(); it++) {
		auto result = std::minmax_element(
			alt.begin(), alt.end(),
			[it](Hierarchy<VariablesType, WeightType>::Alternative *a,
			     Hierarchy<VariablesType, WeightType>::Alternative *b) {
			auto ra = a->getResource();
			auto rb = b->getResource();
			if (ra->mInt.count(*it) > 0) {
			        return ra->mInt[*it] < rb->mInt[*it];
			} else {
			        return ra->mWeight[*it] < rb->mWeight[*it];
			}
		});
		min = max = 0;
		if ((*result.first)->getResource()->mInt.count(*it) > 0) {
			min = (*result.first)->getResource()->mInt[*it];
			max = (*result.second)->getResource()->mInt[*it];
		} else {
			min = (*result.first)->getResource()->mWeight[*it];
			max = (*result.second)->getResource()->mWeight[*it];
		}
		resultValues[*it] = (max - min);
		h_resources.push_back(max-min);
		if (resultValues[*it] == 0) {
			resultValues[*it] = 1;
			h_resources[h_resources.size()-1]=1;
		} else {
			resultValues[*it] /= 9.0;
			h_resources[h_resources.size()-1]=1/9.0;
		}
	}
	// At this point, all the integers and float/WeightType resources  has
	// the max and min values discovered.
	//Prepare the variables to send to the GPU Kernel.
	//Create one vector that get all the map keys of the Data
	//This vector will be represented by
	// V=[key,str(value),key,str(value),...]. All the values are converted to std::string and in the kernel map construction their type are rebuild.
	//To help with the Vector type, use tree types of index, 0 int, 1 float, 2 bool.
	std::string data;//to send vectors to kernel, you must only send the address of the first vector element.
	int dataBytes=0;
	std::vector<int> type;
	std::vector<int> index;
	index.push_back(0);
	int totalResources=0;
	//Iterate through all the alternatives and get their resources.
	for(auto it = alt.begin(); it !=alt.end(); it++) {
		auto resource = (*it)->getResource();
		for(auto const& elem : resource->mInt) {
			//data.push_back(elem.first.c_str());
			int b = std::to_string(elem.second).size();
			data+=std::to_string(elem.second);
			type.push_back(0);
			dataBytes+=b;
			index.push_back(dataBytes);
			totalResources++;
		}
		for(auto const& elem : resource->mWeight) {
			// data.push_back(elem.first.c_str());
			int b = std::to_string(elem.second).size();
			data+=std::to_string(elem.second);
			type.push_back(1);
			dataBytes+=b;
			index.push_back(dataBytes);
			totalResources++;
		}
		for(auto const& elem : resource->mBool) {
			// data.push_back(elem.first.c_str());
			int b = std::to_string(elem.second).size();
			data+=std::to_string(elem.second);
			type.push_back(2);
			dataBytes+=b;
			index.push_back(dataBytes);
			totalResources++;
		}
	}
	totalResources/=alt.size();
	long int resourcesSize = alt.size()*alt.size()*totalResources;
	// totalResources=data.size()/(alt.size()*2);
	std::vector<float> c_result(resourcesSize);
	//All the host data are allocated
	//h_resources
	//Creating the device variables
	dev_array<char> d_data(dataBytes);
	dev_array<int> d_types(type.size());
	dev_array<int> d_index(index.size());
	dev_array<float> d_resources(h_resources.size());
	dev_array<float> d_result(resourcesSize);
	//Alocate the device memory
	//Copy the host variables to device variables
	d_data.set(&data.c_str()[0],dataBytes);
	d_types.set(&type[0],sizeof(int)*type.size());
	d_index.set(&index[0],sizeof(int)*index.size());
	d_resources.set(&h_resources[0], sizeof(float)*h_resources.size());
	//hipMalloc(d_size, alt.size(), sizeof(int), hipMemcpyHostToDevice);
	//hipMalloc(d_sizeCrit, totalResources, sizeof(int), hipMemcpyHostToDevice);

	// setup execution parameters
	// dim3 threadsPerBlock(2,5);
	dim3 threadsPerBlock(block_size,block_size);
	// dim3 numBlocks( 1,1);
	dim3 numBlocks( ceil(alt.size()/(float)threadsPerBlock.x), ceil(alt.size()/(float)threadsPerBlock.y));
	// printf("\nCALLING KERNEL\n");
	// printf("BlockSize: %d. Grid : {%d, %d, %d} blocks. Blocks : {%d, %d, %d} threads.\n", block_size,  numBlocks.x, numBlocks.y, numBlocks.z, threadsPerBlock.x, threadsPerBlock.y, threadsPerBlock.z);
	// acquisitonGKernel<<< grid, threads >>>(d_data.getData(), d_index.getData(), d_types.getData(), d_resources.getData(),d_result.getData(), alt.size(), totalResources);

	acquisitonGKernel<<< numBlocks, threadsPerBlock >>>(d_data.getData(), d_index.getData(), d_types.getData(), d_resources.getData(),d_result.getData(), alt.size(), totalResources);
	hipDeviceSynchronize();
	d_result.get(&c_result[0],resourcesSize);
	hipDeviceSynchronize();
	// std::cout<<"TERMINEI A GPU\n";
	// for(int i=0; i<resourcesSize; i++) {
	//      std::cout<<c_result[i]<<" ";
	// }
	int i=0;
	std::vector<std::vector<std::vector<WeightType> > > allWeights;
	std::vector<std::vector<WeightType> > criteriasWeight;
	std::vector<WeightType> alternativesWeight;
	for (int s=0; s<sheets.size(); s++) {
		criteriasWeight.clear();
		for (int a=0; a<alt.size(); a++) {
			alternativesWeight.clear();
			for (int a2=0; a2<alt.size(); a2++) {
				alternativesWeight.push_back(c_result[i]);
				i++;
			}
			criteriasWeight.push_back(alternativesWeight);
		}
		allWeights.push_back(criteriasWeight);
	}
	// With all the weights calculated, now the weights are set in each edge
	// between the sheets and alternatives
	int aSize = this->hierarchy->getAlternativesCount();
	int size = sheets.size();
	for (int i = 0; i < size; i++) {
		auto edges = sheets[i]->getEdges();
		for (int j = 0; j < aSize; j++) {
			edges[j]->setWeights(allWeights[i][j]);
		}
	}
}

void AHPG::synthesisG() {
	// 1 - Build the construccd the matrix
	buildMatrixG(this->hierarchy->getFocus());
	// printMatrix(this->hierarchy->getFocus());
	// 2 - Normalize the matrix
	buildNormalizedmatrixG(this->hierarchy->getFocus());
	// printNormalizedMatrix(this->hierarchy->getFocus());
	deleteMatrixG(this->hierarchy->getFocus());
	// 3 - calculate the PML
	buildPmlG(this->hierarchy->getFocus());
	deleteNormalizedMatrixG(this->hierarchy->getFocus());
	// printPml(this->hierarchy->getFocus());
	// 4 - calculate the PG
	buildPgG(this->hierarchy->getFocus());
	// printPg(this->hierarchy->getFocus());
	// Print all information
}

void AHPG::consistencyG() {
	iterateFuncG(
		&AHPG::checkConsistencyG<Hierarchy<VariablesType, WeightType>::Criteria>,
		hierarchy->getFocus());
}

// void AHPG::run(std::vector<Hierarchy<VariablesType,WeightType>::Alternative*>
// alt){
void AHPG::run(std::vector<Host *> alternatives) {
	if (alternatives.size() == 0) {
		this->conceptionG(true);
	} else {
		Resource *resource = alternatives[0]->getResource();
		for (auto it : resource->mInt) {
			this->hierarchy->addResource(it.first, "int");
		}
		for (auto it : resource->mWeight) {
			this->hierarchy->addResource(it.first, "float");
		}
		for (auto it : resource->mString) {
			this->hierarchy->addResource(it.first, "string");
		}
		for (auto it : resource->mBool) {
			this->hierarchy->addResource(it.first, "bool");
		}
		this->conceptionG(false);
		this->setAlternatives(alternatives);
	}
	this->acquisitionG();
	this->synthesisG();
	// this->consistencyG();
}

std::map<std::string, int> AHPG::getResult() {
	std::map<std::string, int> result;
	WeightType *values = this->hierarchy->getFocus()->getPg();
	std::vector<std::pair<int, WeightType> > alternativesPair;
	for (int i = 0; i < this->hierarchy->getAlternativesCount(); i++) {
		alternativesPair.push_back(std::make_pair(i, values[i]));
	}
	// std::sort(alternativesPair.begin(), alternativesPair.end(),
	//           [](auto &left, auto &right) {
	//      return left.second > right.second;
	// });
	VariablesType name;
	auto alternatives = this->hierarchy->getAlternatives();
	for (unsigned int i = 0; i < (unsigned int)alternativesPair.size(); i++) {
		name = alternatives[alternativesPair[i].first]->getName();
		result[name] = i + 1;
	}
	return result;
}

void AHPG::setAlternatives(std::vector<Host *> alternatives) {
	this->hierarchy->clearAlternatives();
	for (auto it : alternatives) {
		auto a = new Hierarchy<VariablesType, WeightType>::Alternative(it);
		this->hierarchy->addAlternative(a);
	}
	this->hierarchy->addEdgeSheetsAlternatives();
}
