#include "hip/hip_runtime.h"
#include "ahpg.cuh"

AHPG::AHPG() {
	this->hierarchy = NULL;
	IR[3] = 0.5245;
	IR[4] = 0.8815;
	IR[5] = 1.1086;
	IR[6] = 1.1279;
	IR[7] = 1.3417;
	IR[8] = 1.4056;
	IR[9] = 1.4499;
	IR[10] = 1.4854;
	IR[11] = 1.5141;
	IR[12] = 1.5365;
	IR[13] = 1.5551;
	IR[14] = 1.5713;
	IR[15] = 1.5838;
}

AHPG::~AHPG(){
	delete(hierarchy);
}

void AHPG::setHierarchyG(){
	this->hierarchy = new Hierarchy();
}

char* AHPG::strToLowerG(const char* str) {
	int i;
	char* res = (char*) malloc( strlen(str)+1 );
	for(i=0; i<strlen(str); i++) {
		res[i]=tolower(str[i]);
	}
	res[strlen(str)] = '\0';
	return res;
}

void AHPG::updateAlternativesG() {
	this->hierarchy->clearAlternatives();
	rapidjson::SchemaDocument alternativesSchema =
		JSON::generateSchema("multicriteria/json/alternativesSchema.json");
	rapidjson::Document alternativesData =
		JSON::generateDocument("multicriteria/json/alternativesDataDefault.json");
	rapidjson::SchemaValidator alternativesValidator(alternativesSchema);
	if (!alternativesData.Accept(alternativesValidator))
		JSON::jsonError(&alternativesValidator);
	// domParser(&alternativesData, this);
	this->hierarchy->addEdgeSheetsAlternatives();
}

// Recieve a function address to iterate, used in build matrix, normalized, pml
// and pg functions.
template <typename F> void AHPG::iterateFuncG(F f, Node* node) {
	Edge** edges = node->getEdges(); // array of edges
	Node* criteria; // cruteria node
	int i;
	int size = node->getSize();
	for( i=0; i<size; i++ ) {
		criteria = edges[i]->getNode();
		if (criteria != NULL) {
			(this->*f)(criteria);
		}
	}
}

void AHPG::buildMatrixG(Node* node) {
	int i,j;
	int size = node->getSize(); // get the number of edges
	if ( size == 0 ) return;
	float** matrix = (float**) malloc (sizeof(float*) * size);
	for (i = 0; i < size; i++)
		matrix[i] = (float*) malloc (sizeof(float) * size);

	float* weights;

	for (i = 0; i < size; i++) {
		matrix[i][i] = 1;
		weights = (node->getEdges())[i]->getWeights();
		for (j = i + 1; j < size; j++) {
			matrix[i][j] = weights[j];
			matrix[j][i] = 1 / matrix[i][j];
		}
	}

	node->setMatrix(matrix);

	for(i=0; i< size; i++)
		free(matrix[i]);
	free(matrix);

	iterateFuncG(&AHPG::buildMatrixG, node);
}

void AHPG::buildNormalizedmatrixG(Node* node) {
	int i,j;
	int size = node->getSize();
	if ( size == 0 ) return;
	float** matrix = node->getMatrix(), sum = 0;
	float** nMatrix = (float**) malloc (sizeof(float*) * size);
	for (i = 0; i < size; i++)
		nMatrix[i] = (float*) malloc (sizeof(float) * size);
	for (i = 0; i < size; i++) {
		sum = 0;
		for (j = 0; j < size; j++) {
			sum += matrix[j][i];
		}
		for (j = 0; j < size; j++) {
			nMatrix[j][i] = matrix[j][i] / sum;
		}
	}
	node->setNormalizedMatrix(nMatrix);

	for(i=0; i< size; i++)
		free(nMatrix[i]);
	free(nMatrix);

	iterateFuncG(&AHPG::buildNormalizedmatrixG, node);
}

void AHPG::buildPmlG(Node* node) {
	int i,j;
	int size = node->getSize();
	if ( size == 0 ) return;
	float sum = 0;
	float* pml = (float*) malloc (sizeof(float) * size);
	float** matrix = node->getNormalizedMatrix();
	for (i = 0; i < size; i++) {
		sum = 0;
		for (j = 0; j < size; j++) {
			sum += matrix[i][j];
		}
		pml[i] = sum / (float)size;
	}
	node->setPml(pml);
	iterateFuncG(&AHPG::buildPmlG, node);
	free(pml);
}

void AHPG::buildPgG(Node* node) {
	int i;
	int size = this->hierarchy->getAlternativesSize();
	if ( size == 0 ) return;
	float* pg = (float*) malloc (sizeof(float) * size);
	for (i = 0; i < size; i++) {
		pg[i] = partialPgG(node, i);
	}
	node->setPg(pg, size);
	free(pg);
}

WeightType AHPG::partialPgG(Node* node, int alternative) {
	int i;

	Node* criteria;

	Edge** edges= node->getEdges();
	int size = node->getSize();

	float* pml = node->getPml();
	float partial = 0;
	for (i = 0; i < size; i++) {
		criteria = edges[i]->getNode();
		if (criteria != NULL && criteria->getType()!=node_t::ALTERNATIVE) {
			partial += pml[i] * partialPgG(criteria, alternative);
		} else {
			return pml[alternative];
		}
	}
	return partial;
}

void AHPG::deleteMatrixG(Node* node) {
	int i;
	int size = node->getSize();
	float** matrix = node->getMatrix();
	for (i = 0; i < size; i++)
		free(matrix[i]);
	free(matrix);
	matrix = NULL;
	node->setMatrix(NULL);
	iterateFuncG(&AHPG::deleteMatrixG, node);
}

void AHPG::deleteNormalizedMatrixG(Node* node) {
	int i;
	int size = node->getSize();
	float** nMatrix = node->getNormalizedMatrix();
	for (i = 0; i < size; i++)
		free(nMatrix[i]);
	free(nMatrix);
	nMatrix = NULL;
	node->setNormalizedMatrix(NULL);
	iterateFuncG(&AHPG::deleteNormalizedMatrixG, node);
}

void AHPG::deletePmlG(Node* node){
	int size = node->getSize();
	float* pml = node->getPml();
	free(pml);
	pml = NULL;
	node->setPml(NULL);
	iterateFuncG(&AHPG::deletePmlG, node);
}

void AHPG::checkConsistencyG(Node* node) {
	int i, j;
	int size = node->getSize();
	float** matrix = node->getMatrix();
	float* pml = node->getPml();
	float p[size], lambda = 0, RC = 0;
	for (i = 0; i < size; i++) {
		p[i] = 0;
		for (j = 0; j < size; j++) {
			p[i] += pml[j] * matrix[i][j];
		}
		lambda += (p[i] / pml[i]);
	}
	lambda /= size;
	if (IR[size] > 0) {
		RC = (fabs(lambda - size) / (size - 1)) / IR[size];
	} else {
		// according to AlonsoLamata 2006
		// RC = CI/ RI , where
		// CI = (Lambda_max - n ) / (n-1), and
		// RI = (~Lambda_max - n) / (n-1) , so
		// RC = (Lambda_max - n) / (n-1) / (~Lambda_max - n) / (n-1), then
		// RC = (Lambda_max - n) / (~Lambda_max - n), the ~Lambda_max can be
		// calculated through ~Lambda_max = 2.7699*n-4.3513, thus RC = (Lambda_max -
		// n) / (2.7699 * n - 4.3513 - n ), simplifying RC = (Lambda_max - n) /
		// (1.7699 * n - 4.3513)
		RC = (abs(lambda - size) / (1.7699 * size - 4.3513));
	}
	if (RC > 0.1) {
		printf("ERROR: Criteria %s is inconsistent\n", node->getName());
		printf("RC= %lf\n", RC);
		printf("SIZE= %d\n", size);
		printMatrixG(node);
		printNormalizedMatrixG(node);
		printPmlG(node);
		exit(0);
	}
	iterateFuncG(&AHPG::checkConsistencyG, node);
}


void AHPG::printMatrixG(Node* node) {
	int i,j;
	float** matrix = node->getMatrix();
	int tam = node->getSize();
	printf("Matrix of %s\n", node->getName());
	for (i = 0; i < tam; i++) {
		for (j = 0; j < tam; j++) {
			printf("%010lf\t", matrix[i][j]);
		}
		printf("\n");
	}
	printf("\n");
	iterateFuncG(&AHPG::printMatrixG, node);
}

void AHPG::printNormalizedMatrixG(Node* node) {
	int i,j;
	float **matrix = node->getNormalizedMatrix();
	int tam = node->getSize();
	printf("Normalized Matrix of %s\n", node->getName());
	for (i = 0; i < tam; i++) {
		for (j = 0; j < tam; j++) {
			printf("%010lf\t", matrix[i][j]);
		}
		printf("\n");
	}
	std::cout << "\n";
	iterateFuncG(&AHPG::printNormalizedMatrixG, node);
}

void AHPG::printPmlG(Node* node) {
	int i;
	float* pml = node->getPml();
	int tam = node->getSize();
	printf("PML of %s\n", node->getName());
	for (i = 0; i < tam; i++) {
		printf("%010lf\t", pml[i]);
	}
	printf("\n");
	iterateFuncG(&AHPG::printPmlG, node);
}

void AHPG::printPgG(Node* node) {
	int i;
	float* pg = node->getPg();
	int tam = this->hierarchy->getAlternativesSize();
	printf("PG of %s\n", node->getName());
	for (i = 0; i < tam; i++) {
		printf("%010lf\t",pg[i]);
	}
	printf("\n");
}

void AHPG::generateContentSchemaG() {
	std::string names;
	std::string text = "{\"$schema\":\"http://json-schema.org/draft-04/"
	                   "schema#\",\"definitions\": {\"alternative\": {\"type\": "
	                   "\"array\",\"minItems\": 1,\"items\":{\"properties\": {";
	H_Resource* resource = this->hierarchy->getResource();
	int i, size = resource->getDataSize();
	for (i=0; i<size; i++) {
		text += "\"" + std::to_string(resource->getResource(i)) + "\":{\"type\":\"number\"},";
		names += "\"" + std::string(resource->getResourceName(i)) + "\",";
	}
	names.pop_back();
	text.pop_back();
	text += "},\"additionalProperties\": false,\"required\": [" + names +
	        "]}}},\"type\": \"object\",\"minProperties\": "
	        "1,\"additionalProperties\": false,\"properties\": "
	        "{\"alternatives\": {\"$ref\": \"#/definitions/alternative\"}}}";
	JSON::writeJson("multicriteria/json/alternativesSchema.json", text);
}

void AHPG::resourcesParserG(genericValue* dataResource) {
	std::string variableName, variableType;
	for (auto &arrayData : dataResource->value.GetArray()) {
		variableName = variableType = "";
		for (auto &objectData : arrayData.GetObject()) {
			if (strcmp(objectData.name.GetString(), "name") == 0) {
				variableName = objectData.value.GetString();
			} else if (strcmp(objectData.name.GetString(), "variableType") == 0) {
				variableType = strToLowerG(objectData.value.GetString());
			} else {
				std::cout << "Error in reading resources\nExiting...\n";
				exit(0);
			}
		}
		this->hierarchy->addResource((char*)variableName.c_str());
	}
}

void AHPG::hierarchyParserG(genericValue* dataObjective) {
	for (auto &hierarchyObject : dataObjective->value.GetObject()) {
		if (strcmp(hierarchyObject.name.GetString(), "name") == 0) {
			this->hierarchy->addFocus(strToLowerG(hierarchyObject.value.GetString())); // create the Focus* in the hierarchy;
		} else if (strcmp(hierarchyObject.name.GetString(), "childs") == 0) {
			criteriasParserG(&hierarchyObject, this->hierarchy->getFocus());
		} else {
			std::cout << "AHP -> Unrecognizable Type\nExiting...\n";
			exit(0);
		}
	}
}

void AHPG::criteriasParserG(genericValue* dataCriteria, Node* parent) {
	char* name;
	bool leaf = false;
	float* weight;
	int index=0;
	for (auto &childArray : dataCriteria->value.GetArray()) {
		weight = NULL;
		for (auto &child : childArray.GetObject()) {
			const char* n = child.name.GetString();
			if (strcmp(n, "name") == 0) {
				name = strToLowerG(child.value.GetString());
			} else if (strcmp(n, "leaf") == 0) {
				leaf = child.value.GetBool();
			} else if (strcmp(n, "weight") == 0) {
				for (auto &weightChild : child.value.GetArray()) {
					weight[index]=weightChild.GetFloat();
					index++;
				}
			} else if (strcmp(n, "childs") == 0) {
				// at this point, all the criteria variables were read, now the document
				// has the child's of the criteria. To put the childs corretly inside
				// the hierarchy, the criteria node has to be created.
				auto criteria = this->hierarchy->addCriteria(name);
				criteria->setLeaf(leaf);
				this->hierarchy->addEdge(parent, criteria, weight);
				// with the criteria node added, the call recursively the
				// criteriasParser.
				criteriasParserG(&child, criteria);
			}
		}
		if (leaf) {
			auto criteria = this->hierarchy->addCriteria(name);
			criteria->setLeaf(leaf);
			this->hierarchy->addSheets(criteria);
			this->hierarchy->addEdge(parent, criteria, weight);
		}
	}
}

void AHPG::alternativesParserG(genericValue* dataAlternative) {
	for (auto &arrayAlternative : dataAlternative->value.GetArray()) {
		auto alternative = this->hierarchy->addAlternative();
		for (auto &alt : arrayAlternative.GetObject()) {
			std::string name(alt.name.GetString());
			if (alt.value.IsNumber()) {
				alternative->getResource()->addResource((char*) name.c_str(), alt.value.GetFloat());
			} else if (alt.value.IsBool()) {
				bool b = alt.value.GetBool();
				if(b==true) {
					alternative->getResource()->addResource((char*) name.c_str(), 1);
				}else{
					alternative->getResource()->addResource((char*) name.c_str(), 0);
				}
			} else {
				if(name=="name") {
					alternative->setName(strToLowerG(alt.value.GetString()));
				}
			}
		}
	}
}

void AHPG::domParserG(rapidjson::Document *data) {
	for (auto &m : data->GetObject()) { // query through all objects in data.
		if (strcmp(m.name.GetString(), "resources") == 0) {
			resourcesParserG(&m);
		} else if (strcmp(m.name.GetString(), "objective") == 0) {
			hierarchyParserG(&m);
		} else if (strcmp(m.name.GetString(), "alternatives") == 0) {
			alternativesParserG(&m);
		}
	}
}

void AHPG::conceptionG(bool alternativeParser) {
	// The hierarchy contruction were divided in three parts, first the resources
	// file was to be loaded to construct the alternatives dynamically. Second the
	// hierarchy focus and criteria were loaded in the hierarchyData.json, and
	// finally the alternatives were loaded.
	if (alternativeParser) {
		rapidjson::SchemaDocument resourcesSchema =
			JSON::generateSchema("multicriteria/json/resourcesSchema.json");
		rapidjson::Document resourcesData =
			JSON::generateDocument("multicriteria/json/resourcesData.json");
		rapidjson::SchemaValidator resourcesValidator(resourcesSchema);
		if (!resourcesData.Accept(resourcesValidator))
			JSON::jsonError(&resourcesValidator);
		domParserG(&resourcesData);
		generateContentSchemaG();
	}
	// After reading the resoucesData, new alternativesSchema has to be created.
	// Parser the Json File that contains the Hierarchy
	rapidjson::SchemaDocument hierarchySchema =
		JSON::generateSchema("multicriteria/json/hierarchySchema.json");
	rapidjson::Document hierarchyData =
		JSON::generateDocument("multicriteria/json/hierarchyData.json");
	rapidjson::SchemaValidator hierarchyValidator(hierarchySchema);
	if (!hierarchyData.Accept(hierarchyValidator))
		JSON::jsonError(&hierarchyValidator);
	domParserG(&hierarchyData);
	if (alternativeParser) {
		// The Json Data is valid and can be used to construct the hierarchy.
		rapidjson::SchemaDocument alternativesSchema =
			JSON::generateSchema("multicriteria/json/alternativesSchema.json");
		rapidjson::Document alternativesData = JSON::generateDocument(
			"multicriteria/json/alternativesDataDefault.json");
		rapidjson::SchemaValidator alternativesValidator(alternativesSchema);
		if (!alternativesData.Accept(alternativesValidator))
			JSON::jsonError(&alternativesValidator);
		domParserG(&alternativesData);
		this->hierarchy->addEdgeSheetsAlternatives();
	}
}

/*
    This function is used to make the comparison between the alternatives.
    Data represents all the data in the alternatives.
    Types represents the types of the alternative data.
    Max_mix represents to normalize the values.
    cmp is the result array of the comparison.
    size is the amount of the alternatives in the hierarchy.
    sizeCrit represent the size of sheets, used to jump through the data and types array.
 */

//QUEBRAR EM TIES PARA USAR MEMORIA COMPARTILHADA
__global__
void acquisitonGKernel(char * data, int* index,  int* types, float* max_min, float* cmp, int size, int sizeCrit){
	// //The row is the number of the alternative
	// int row = blockIdx.x*size+threadIdx.x;
	// //The col is the number of the criteria
	// int col = blockIdx.y*size+threadIdx.y;
	// int value_alt1_int, value_alt2_int, t;
	// float value_alt1_float, value_alt2_float;
	// char* sub,*sub2;
	// // int k=0;
	// // for(int i=0; data[i]!='\0'; i++) {
	// //      if(i==index[k]) {printf("|"); k++;}
	// //      printf("%c",data[i]);
	// // }
	// // printf("ALT SIZE: %d , CRIT SIZE %d\n",size, sizeCrit);
	// // if(row==0 && col <sizeCrit) { //the thread can do the work
	// if(row<size && col <sizeCrit) {                 //the thread can do the work
	//      int indexRead = row*sizeCrit+col;
	//      // printf("NEW THREAD ROW %d COL %d SIZE %d SIZECRIT %d\n",row,col,size,sizeCrit);
	//      value_alt1_int=0;
	//      value_alt1_float=0.0f;
	//      t=0;
	//      // printf("%d # %d # %d # %d\n",indexRead, indexRead+1,index[indexRead],index[indexRead+1]);
	//      sub=copyStr(data,index[indexRead],index[indexRead+1]);
	//      if(types[row*sizeCrit+col]==0 || types[row*sizeCrit+col]==2) {
	//              value_alt1_int=char_to_int(sub);
	//              // printf("CONVERTED INT %d\n",value_alt1_int);
	//      }else if(types[row*sizeCrit+col]==1) {
	//              value_alt1_float=char_to_float(sub);
	//              // printf("CONVERTED FLOAT %f\n",value_alt1_float);
	//      }
	//      for(int alt=0; alt<size; alt++) {
	//              sub2=copyStr(data,index[alt*sizeCrit+col],index[alt*sizeCrit+(col+1)]);
	//              // printf("ALTERNATIVE %d - %s # %s\n",alt,sub,sub2);
	//              value_alt2_int=0;
	//              value_alt2_float=0.0f;
	//              //alt*sizeCrit+col will jump over the alternatives to get the same coleria value.
	//              if(types[alt*sizeCrit+col]==0) {
	//                      t=0;
	//                      value_alt2_int=char_to_int(sub2);
	//              }else if(types[alt*sizeCrit+col]==1) {
	//                      t=1;
	//                      value_alt2_float=char_to_float(sub2);
	//              }else if(types[alt*sizeCrit+col]==2) {
	//                      t=2;
	//                      value_alt2_int=char_to_int(sub2);
	//              }
	//              // printf("DIVIDED BY %f\n",max_min[col]);
	//              // printf("SIZE: %d\n",size);
	//              int indexWrite = row*size*sizeCrit+col*size+alt;
	//              // int indexWrite = row*size*size/2+alt;
	//              // printf("WERE I WRITE %d\n",row*size*sizeCrit+col*size+alt);
	//              //Its used row*size*size/2 to jump correctly in the vector and set the values
	//              if(t==0) {
	//                      value_alt1_int==value_alt2_int ? cmp[indexWrite]=1 : cmp[indexWrite] = (value_alt1_int-value_alt2_int) / (float) max_min[col];
	//                      // printf("Write in T0 %f\n",cmp[indexWrite]);
	//              }else if(t==1) {
	//                      // if(value_alt1_float!=value_alt2_float) printf("DIF %f %f\n",value_alt1_float,value_alt2_float);
	//                      value_alt1_float==value_alt2_float ? cmp[indexWrite]=1 : cmp[indexWrite] = (value_alt1_float - value_alt2_float) / (float) max_min[col];
	//                      // printf("Write in T1 %f\n",cmp[indexWrite]);
	//              }
	//              else if(t==2) {
	//                      // printf("ALTERNATIVE %d - %s # %s\n",alt,sub,sub2);
	//                      // printf("BOOL %d %d\n",value_alt1_int,value_alt2_int);
	//                      if(value_alt1_int==value_alt2_int) cmp[indexWrite]=1;
	//                      else if(value_alt1_int==1) cmp[indexWrite]=9;
	//                      else if(value_alt1_int==0) cmp[indexWrite]=1/9.0f;
	//                      // printf("Write in T2 %f\n", cmp[indexWrite]);
	//              }else{
	//                      printf("UNESPECTED VALUE FOR T\n");
	//              }
	//      }
	// }
}

void AHPG::acquisitionG() {
	// //Get the device info
	// int devID;
	// hipDeviceProp_t props;
	// hipGetDevice(&devID);
	// hipGetDeviceProperties(&props, devID);
	// int block_size = (props.major < 2) ? 16 : 32;
	//
	// // Para gerar os pesos das alterntivas, será primeiro captado o MIN e MAX
	// // valor das alternativas , após isso será montada as matrizes de cada sheet
	// auto alt = this->hierarchy->getAlternatives();
	// auto sheets = this->hierarchy->getSheets();
	// std::vector<std::string> sheetsNames;
	// for_each(sheets.begin(), sheets.end(),
	//          [&sheetsNames,
	//           this](Hierarchy<VariablesType, WeightType>::Criteria *c) mutable {
	//      // Get the name off all sheet nodes that aren't boolean
	//      if (this->hierarchy->getResource()->mBool.count(c->getName()) ==
	//          0) {
	//              sheetsNames.push_back(c->getName());
	//      }
	// });
	// std::map<std::string, float> resultValues;
	// std::vector<float>h_resources;
	// float min, max;
	// for (auto it = sheetsNames.begin(); it != sheetsNames.end(); it++) {
	//      auto result = std::minmax_element(
	//              alt.begin(), alt.end(),
	//              [it](Hierarchy<VariablesType, WeightType>::Alternative *a,
	//                   Hierarchy<VariablesType, WeightType>::Alternative *b) {
	//              auto ra = a->getResource();
	//              auto rb = b->getResource();
	//              if (ra->mInt.count(*it) > 0) {
	//                      return ra->mInt[*it] < rb->mInt[*it];
	//              } else {
	//                      return ra->mWeight[*it] < rb->mWeight[*it];
	//              }
	//      });
	//      min = max = 0;
	//      if ((*result.first)->getResource()->mInt.count(*it) > 0) {
	//              min = (*result.first)->getResource()->mInt[*it];
	//              max = (*result.second)->getResource()->mInt[*it];
	//      } else {
	//              min = (*result.first)->getResource()->mWeight[*it];
	//              max = (*result.second)->getResource()->mWeight[*it];
	//      }
	//      resultValues[*it] = (max - min);
	//      h_resources.push_back(max-min);
	//      if (resultValues[*it] == 0) {
	//              resultValues[*it] = 1;
	//              h_resources[h_resources.size()-1]=1;
	//      } else {
	//              resultValues[*it] /= 9.0;
	//              h_resources[h_resources.size()-1]=1/9.0;
	//      }
	// }
	//
	// resultValues.clear();
	//
	// // At this point, all the integers and float/WeightType resources  has
	// // the max and min values discovered.
	// //Prepare the variables to send to the GPU Kernel.
	// //Create one vector that get all the map keys of the Data
	// //This vector will be represented by
	// // V=[key,str(value),key,str(value),...]. All the values are converted to std::string and in the kernel map construction their type are rebuild.
	// //To help with the Vector type, use tree types of index, 0 int, 1 float, 2 bool.
	// std::string data;//to send vectors to kernel, you must only send the address of the first vector element.
	// int dataBytes=0;
	// std::vector<int> type;
	// std::vector<int> index;
	// index.push_back(0);
	// int totalResources=0;
	// //Iterate through all the alternatives and get their resources.
	// for(auto it = alt.begin(); it !=alt.end(); it++) {
	//      auto resource = (*it)->getResource();
	//      for(auto const& elem : resource->mInt) {
	//              //data.push_back(elem.first.c_str());
	//              int b = std::to_string(elem.second).size();
	//              data+=std::to_string(elem.second);
	//              type.push_back(0);
	//              dataBytes+=b;
	//              index.push_back(dataBytes);
	//              totalResources++;
	//      }
	//      for(auto const& elem : resource->mWeight) {
	//              // data.push_back(elem.first.c_str());
	//              int b = std::to_string(elem.second).size();
	//              data+=std::to_string(elem.second);
	//              type.push_back(1);
	//              dataBytes+=b;
	//              index.push_back(dataBytes);
	//              totalResources++;
	//      }
	//      for(auto const& elem : resource->mBool) {
	//              // data.push_back(elem.first.c_str());
	//              int b = std::to_string(elem.second).size();
	//              data+=std::to_string(elem.second);
	//              type.push_back(2);
	//              dataBytes+=b;
	//              index.push_back(dataBytes);
	//              totalResources++;
	//      }
	// }
	// totalResources/=alt.size();
	// long int resourcesSize = alt.size()*alt.size()*totalResources;
	// // totalResources=data.size()/(alt.size()*2);
	// std::vector<float> c_result(resourcesSize);
	// //All the host data are allocated
	// //h_resources
	// //Creating the device variables
	// dev_array<char> d_data(dataBytes);
	// dev_array<int> d_types(type.size());
	// dev_array<int> d_index(index.size());
	// dev_array<float> d_resources(h_resources.size());
	// dev_array<float> d_result(resourcesSize);
	// //Alocate the device memory
	// //Copy the host variables to device variables
	// d_data.set(&data.c_str()[0],dataBytes);
	// d_types.set(&type[0],sizeof(int)*type.size());
	// d_index.set(&index[0],sizeof(int)*index.size());
	// d_resources.set(&h_resources[0], sizeof(float)*h_resources.size());
	// //hipMalloc(d_size, alt.size(), sizeof(int), hipMemcpyHostToDevice);
	// //hipMalloc(d_sizeCrit, totalResources, sizeof(int), hipMemcpyHostToDevice);
	//
	// //Uma vez os valores copiados para o cuda eles podem ser deletados
	// data="";
	// type.clear();
	// type.shrink_to_fit();
	// index.clear();
	// index.shrink_to_fit();
	//
	// // setup execution parameters
	// // dim3 threadsPerBlock(2,5);
	// dim3 threadsPerBlock(block_size,block_size);
	// // dim3 numBlocks( 1,1);
	// dim3 numBlocks( ceil(alt.size()/(float)threadsPerBlock.x), ceil(alt.size()/(float)threadsPerBlock.y));
	// // printf("\nCALLING KERNEL\n");
	// // printf("BlockSize: %d. Grid : {%d, %d, %d} blocks. Blocks : {%d, %d, %d} threads.\n", block_size,  numBlocks.x, numBlocks.y, numBlocks.z, threadsPerBlock.x, threadsPerBlock.y, threadsPerBlock.z);
	// // acquisitonGKernel<<< grid, threads >>>(d_data.getData(), d_index.getData(), d_types.getData(), d_resources.getData(),d_result.getData(), alt.size(), totalResources);
	//
	// acquisitonGKernel<<< numBlocks, threadsPerBlock >>>(d_data.getData(), d_index.getData(), d_types.getData(), d_resources.getData(),d_result.getData(), alt.size(), totalResources);
	// hipDeviceSynchronize();
	// d_result.get(&c_result[0],resourcesSize);
	// hipDeviceSynchronize();
	// d_data.resize(0);
	// d_types.resize(0);
	// d_index.resize(0);
	// d_resources.resize(0);
	// d_result.resize(0);
	// // std::cout<<"TERMINEI A GPU\n";
	// // for(int i=0; i<resourcesSize; i++) {
	// //      std::cout<<c_result[i]<<" ";
	// // }
	// int i=0;
	// std::vector<std::vector<std::vector<WeightType> > > allWeights;
	// std::vector<std::vector<WeightType> > criteriasWeight;
	// std::vector<WeightType> alternativesWeight;
	// for (int s=0; s<sheets.size(); s++) {
	//      for (int a=0; a<alt.size(); a++) {
	//              alternativesWeight.clear();
	//              alternativesWeight.shrink_to_fit();
	//              for (int a2=0; a2<alt.size(); a2++) {
	//                      alternativesWeight.push_back(c_result[i++]);
	//              }
	//              criteriasWeight.push_back(alternativesWeight);
	//      }
	//      allWeights.push_back(criteriasWeight);
	//      criteriasWeight[0].clear();
	//      criteriasWeight[0].shrink_to_fit();
	//      criteriasWeight.clear();
	//      criteriasWeight.shrink_to_fit();
	// }
	// c_result.clear();
	// c_result.shrink_to_fit();
	// alternativesWeight.clear();
	// alternativesWeight.shrink_to_fit();
	// criteriasWeight.clear();
	// criteriasWeight.shrink_to_fit();
	// // With all the weights calculated, now the weights are set in each edge
	// // between the sheets and alternatives
	// int aSize = this->hierarchy->getAlternativesCount();
	// int size = sheets.size();
	// for (int i = 0; i < size; i++) {
	//      auto edges = sheets[i]->getEdges();
	//      for (int j = 0; j < aSize; j++) {
	//              edges[j]->setWeights(allWeights[i][j]);
	//              allWeights[i][j].clear();
	//              allWeights[i][j].shrink_to_fit();
	//      }
	//      allWeights[i].clear();
	//      allWeights[i].shrink_to_fit();
	// }
	//
	// allWeights.clear();
	// allWeights.shrink_to_fit();
}

void AHPG::synthesisG() {
	// 1 - Build the construccd the matrix
	buildMatrixG(this->hierarchy->getFocus());
	// printMatrix(this->hierarchy->getFocus());
	// 2 - Normalize the matrix
	buildNormalizedmatrixG(this->hierarchy->getFocus());
	// printNormalizedMatrix(this->hierarchy->getFocus());
	deleteMatrixG(this->hierarchy->getFocus());
	// 3 - calculate the PML
	buildPmlG(this->hierarchy->getFocus());
	deleteNormalizedMatrixG(this->hierarchy->getFocus());
	// printPml(this->hierarchy->getFocus());
	// 4 - calculate the PG
	buildPgG(this->hierarchy->getFocus());
	// printPg(this->hierarchy->getFocus());
	// Print all information
}

void AHPG::consistencyG() {
	iterateFuncG( &AHPG::checkConsistencyG, hierarchy->getFocus() );

}

// void AHPG::run(std::vector<Hierarchy<VariablesType,WeightType>::Alternative*>
// alt){
void AHPG::run(Host** alternatives, int size) {
	if (size == 0) {
		this->conceptionG(true);
	} else {
		// this->hierarchy->clearAlternatives(); // made in the setAlternatives function
		this->hierarchy->clearResource();

		Resource *resource = alternatives[0]->getResource();

		for (auto it : resource->mInt) {
			this->hierarchy->addResource((char*)it.first.c_str());
		}
		for (auto it : resource->mWeight) {
			this->hierarchy->addResource((char*)it.first.c_str());
		}
		for (auto it : resource->mBool) {
			this->hierarchy->addResource((char*)it.first.c_str());
		}
		this->conceptionG(false);
		this->setAlternatives(alternatives, size);
	}
	this->acquisitionG();
	this->synthesisG();
	// this->consistency();
}

std::map<int,char*> AHPG::getResult() {
	std::map<int,char*> result;
	float* values = this->hierarchy->getFocus()->getPg();
	std::vector<std::pair<int, float> > alternativesPair;

	unsigned int i;

	for (i = 0; i < this->hierarchy->getAlternativesSize(); i++) {
		alternativesPair.push_back(std::make_pair(i, values[i]));
	}
	// Nao e necessario fazer sort, o map ja realiza o sort do map pela chave em ordem acendente (menor - maior)
	// std::sort(alternativesPair.begin(), alternativesPair.end(),
	//           [](auto &left, auto &right) {
	//      return left.second > right.second;
	// });

	char* name;

	auto alternatives = this->hierarchy->getAlternatives();
	for (i = 0; i < (unsigned int)alternativesPair.size(); i++) {
		name = alternatives[alternativesPair[i].first]->getName();
		result[i+1] = name;
	}
	return result;
}

void AHPG::setAlternatives(Host** alternatives, int size) {
	int i;

	this->hierarchy->clearAlternatives();

	Resource* resource;

	for ( i=0; i<size; i++) {
		resource = alternatives[i]->getResource(); // Host resource

		Node* a = new Node(); // create the new node

		a->setResource(*this->hierarchy->getResource()); // set the default resources in the node

		a->setName((char*) alternatives[i]->getName().c_str()); // set the node name

		// Update the node h_resource values by the host resource values
		for (auto it : resource->mInt) {
			a->setResource((char*)it.first.c_str(), (float) it.second);
		}
		for (auto it : resource->mWeight) {
			a->setResource((char*)it.first.c_str(), it.second);
		}
		for (auto it : resource->mBool) {
			a->setResource((char*)it.first.c_str(), (float) it.second);
		}

		this->hierarchy->addAlternative(a);
	}

	this->hierarchy->addEdgeSheetsAlternatives();
}
